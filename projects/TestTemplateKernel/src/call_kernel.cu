

#include <hip/hip_runtime.h>

#include <kernel.cuh>


#define ThreadsPerBlock 128
template<typename T>
__host__ void call_kernel_power(T *base, T *n, T *output, T elementCount) {


    int *dev_base, *dev_n, *dev_output;
    int gridX = (elementCount+ThreadsPerBlock-1)/ThreadsPerBlock;

   hipMalloc( (void**)&dev_base, elementCount * sizeof(int) );
	hipMalloc( (void**)&dev_n, elementCount * sizeof(int) );
	hipMalloc( (void**)&dev_output, elementCount * sizeof(int) );

   hipMemcpy( dev_base, base, elementCount * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy( dev_n, n, elementCount * sizeof(int), hipMemcpyHostToDevice);

	power<T><<<gridX,ThreadsPerBlock>>>(dev_base, dev_n, dev_output, elementCount);

    hipMemcpy( output, dev_output, elementCount * sizeof(int), hipMemcpyDeviceToHost);

	hipFree( dev_base );
	hipFree( dev_n );
	hipFree( dev_output );
}


// explicit instantiation
template __host__ void call_kernel_power(int *base, int *n, int *output, int elementCount);

