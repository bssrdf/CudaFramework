#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include "TypenameComparision.hpp"
#include "CudaFramework/CudaModern/CudaError.hpp"

static char *sSDKsample = "CmakeTestSimple";


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
__global__ void test_kernel(float * c, float * a, float * b);
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	printf("[ %s ]\n", sSDKsample);


	runTest(argc, argv);
	system("pause");
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv)
{

	// =========================== WITHOUT CUUTIL

	hipSetDevice(0);

	int devID;
	hipDeviceProp_t props;

	// get number of SMs on this GPU
	hipGetDevice(&devID);
	(hipGetDeviceProperties(&props, devID));

	printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);


	// allocate device memory
	float* d_A;
	float* A = (float*) malloc(sizeof(float));
	*A = 20;
   printf("A is: %f",*A);
	CHECK_CUDA(hipMalloc((void**) &d_A, sizeof(float)));
	float* d_B;
	float* B = (float*) malloc(sizeof(float));
	*B = 10;
   printf("A is: %f",*B);
	CHECK_CUDA(hipMalloc((void**) &d_B, sizeof(float)));

	// copy host memory to device
	CHECK_CUDA(hipMemcpy(d_A, A, sizeof(float),hipMemcpyHostToDevice) );
	CHECK_CUDA(hipMemcpy(d_B, B, sizeof(float),hipMemcpyHostToDevice) );

	// allocate device memory for result
	float* d_C;
	CHECK_CUDA(hipMalloc((void**) &d_C, sizeof(float)));

	// allocate host memory for the result
	float* C = (float*) malloc(sizeof(float));
   *C = 0;

	test_kernel<<< 1, 1 >>>(d_C, d_A, d_B);
   hipDeviceSynchronize();
	CHECK_CUDA(hipMemcpy(C, d_C, sizeof(float), hipMemcpyDeviceToHost) );

	printf("Result is: %f",*C);

	// check if kernel execution generated and error
	//cutilCheckMsg("Kernel execution failed");


	// clean up memory
	free(A);
	free(B);
	free(C);
	(hipFree(d_A));
	(hipFree(d_B));
	(hipFree(d_C));

	hipDeviceReset();


}


__global__
void test_kernel(float * c, float * a, float * b){
		*c = *a + *b;
}
