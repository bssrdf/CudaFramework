// ========================================================================================
//  CudaFramework
//  Copyright (C) 2014 by Gabriel Nützi <nuetzig (at) imes (d0t) mavt (d0t) ethz (d0t) ch>
//
//  This Source Code Form is subject to the terms of the GNU GPL 3.0 licence. 
//  If a copy of the GNU GPL 3.0 was not distributed with this
//  file, you can obtain one at http://opensource.org/licenses/GPL-3.0.
// ========================================================================================


#include <hip/hip_runtime.h>


namespace utilCuda{

    __global__ void kernelVersionShim(){};

    __host__ hipError_t getCudaCompilerVersion(unsigned int ordinal, hipFuncAttributes & attr){
        hipSetDevice(ordinal);
        void (*p1)(void) = kernelVersionShim;
        const char * p2 = (char*)p1;
        return hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(p2));
    }

};
