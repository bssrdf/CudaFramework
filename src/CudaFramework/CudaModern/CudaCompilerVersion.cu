
#include <hip/hip_runtime.h>


namespace utilCuda{

    __global__ void kernelVersionShim(){};

    __host__ hipError_t getCudaCompilerVersion(unsigned int ordinal, hipFuncAttributes & attr){
        hipSetDevice(ordinal);
        void (*p1)(void) = kernelVersionShim;
        const char * p2 = (char*)p1;
        return hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(p2));
    }

};
