#include <hip/hip_runtime.h>
#include <ConvergenceCheck.cuh>

#include "CudaFramework/CudaModern/CudaMatrix.hpp"

#define PREC double
template __host__ void  ConvCheck::convCheckKernelWrap<false,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfContacts,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ConvCheck::convCheckKernelWrap<false,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfContacts,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

// Explicit code generation for the CUDA compile side (for all template parameter)
#define PREC double
template __host__ void  ConvCheck::convCheckKernelWrap<true,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfBodies,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ConvCheck::convCheckKernelWrap<true,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfBodies,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC
