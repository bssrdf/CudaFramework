// ========================================================================================
//  CudaFramework
//  Copyright (C) 2014 by Gabriel Nützi <nuetzig (at) imes (d0t) mavt (d0t) ethz (d0t) ch>
//
//  This Source Code Form is subject to the terms of the GNU GPL 3.0 licence. 
//  If a copy of the GNU GPL 3.0 was not distributed with this
//  file, you can obtain one at http://opensource.org/licenses/GPL-3.0.
// ========================================================================================

#include <hip/hip_runtime.h>
#include <ConvergenceCheck.cuh>

#include "CudaFramework/CudaModern/CudaMatrix.hpp"

#define PREC double
template __host__ void  ConvCheck::convCheckKernelWrap<false,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfContacts,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ConvCheck::convCheckKernelWrap<false,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfContacts,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

// Explicit code generation for the CUDA compile side (for all template parameter)
#define PREC double
template __host__ void  ConvCheck::convCheckKernelWrap<true,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfBodies,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ConvCheck::convCheckKernelWrap<true,true,PREC>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                  utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                  utilCuda::CudaMatrix<PREC> outputBuffer,
                                                  PREC* redBufferIn,
                                                  unsigned int numberOfBodies,
                                                  VariantLaunchSettings variantSettings,
                                                  PREC relTol,
                                                  PREC absTol);
#undef PREC
