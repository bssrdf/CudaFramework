#include <hip/hip_runtime.h>

#include "ContactInit.cuh"

#include "CudaMatrix.hpp"
#include "VariantLaunchSettings.hpp"

#define PREC double
template __host__ void  ContactInit::contactInitKernelWrap<false>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                     utilCuda::CudaMatrix<PREC> contactBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> indexBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                     utilCuda::CudaMatrix<PREC> outputBuffer,
                                                     unsigned int numberOfContacts,
                                                     VariantLaunchSettings variantSettings);
#undef PREC

#define PREC float
template __host__ void  ContactInit::contactInitKernelWrap<false>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                     utilCuda::CudaMatrix<PREC> contactBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> indexBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                     utilCuda::CudaMatrix<PREC> outputBuffer,
                                                     unsigned int numberOfContacts,
                                                     VariantLaunchSettings variantSettings);
#undef PREC
// Explicit code generation for the CUDA compile side (for all template parameter)
#define PREC double
template __host__ void  ContactInit::contactInitKernelWrap<true>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                     utilCuda::CudaMatrix<PREC> contactBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> indexBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                     utilCuda::CudaMatrix<PREC> outputBuffer,
                                                     unsigned int numberOfContacts,
                                                     VariantLaunchSettings variantSettings);
#undef PREC

#define PREC float
template __host__ void  ContactInit::contactInitKernelWrap<true>(utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                     utilCuda::CudaMatrix<PREC> contactBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> indexBuffer,
                                                     utilCuda::CudaMatrix<unsigned int> globalBuffer,
                                                     utilCuda::CudaMatrix<PREC> outputBuffer,
                                                     unsigned int numberOfContacts,
                                                     VariantLaunchSettings variantSettings);
#undef PREC


