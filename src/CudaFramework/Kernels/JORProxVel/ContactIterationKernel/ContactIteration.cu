// ========================================================================================
//  CudaFramework
//  Copyright (C) 2014 by Gabriel Nützi <nuetzig (at) imes (d0t) mavt (d0t) ethz (d0t) ch>
//
//  This Source Code Form is subject to the terms of the GNU GPL 3.0 licence. 
//  If a copy of the GNU GPL 3.0 was not distributed with this
//  file, you can obtain one at http://opensource.org/licenses/GPL-3.0.
// ========================================================================================

#include <hip/hip_runtime.h>
#include <ContactIteration.cuh>

#include "CudaFramework/CudaModern/CudaMatrix.hpp"

#define PREC double
template __host__ void  ContIter::contIterKernelWrap<false,true>( utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                        utilCuda::CudaMatrix<PREC> contactBuffer,
                                                        utilCuda::CudaMatrix<unsigned> globalBuffer,
                                                        PREC * reductionBuffer,
                                                        utilCuda::CudaMatrix<unsigned int> indexSetBuffer,
                                                        utilCuda::CudaMatrix<PREC> outputBuffer,
                                                        unsigned int numberOfContacts,
                                                        VariantLaunchSettings variantSettings,
                                                        unsigned int totalRedNumber,
                                                        PREC relTol,
                                                        PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ContIter::contIterKernelWrap<false,true>( utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                        utilCuda::CudaMatrix<PREC> contactBuffer,
                                                        utilCuda::CudaMatrix<unsigned> globalBuffer,
                                                        PREC* reductionBuffer,
                                                        utilCuda::CudaMatrix<unsigned int> indexSetBuffer,
                                                        utilCuda::CudaMatrix<PREC> outputBuffer,
                                                        unsigned int numberOfContacts,
                                                        VariantLaunchSettings variantSettings,
                                                        unsigned int totalRedNumber,
                                                        PREC relTol,
                                                        PREC absTol);
#undef PREC


// Explicit code generation for the CUDA compile side (for all template parameter)
#define PREC double
template __host__ void  ContIter::contIterKernelWrap<true,true>(  utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                        utilCuda::CudaMatrix<PREC> contactBuffer,
                                                        utilCuda::CudaMatrix<unsigned> globalBuffer,
                                                        PREC * m_reductionBuffer,
                                                        utilCuda::CudaMatrix<unsigned int> indexSetBuffer,
                                                        utilCuda::CudaMatrix<PREC> outputBuffer,
                                                        unsigned int numberOfContacts,
                                                        VariantLaunchSettings variantSettings,
                                                        unsigned int totalRedNumber,
                                                        PREC relTol,
                                                        PREC absTol);
#undef PREC

#define PREC float
template __host__ void  ContIter::contIterKernelWrap<true,true>( utilCuda::CudaMatrix<PREC> bodyBuffer,
                                                            utilCuda::CudaMatrix<PREC> contactBuffer,
                                                            utilCuda::CudaMatrix<unsigned> globalBuffer,
                                                            PREC * m_reductionBuffer,
                                                            utilCuda::CudaMatrix<unsigned int> indexSetBuffer,
                                                            utilCuda::CudaMatrix<PREC> outputBuffer,
                                                            unsigned int numberOfContacts,
                                                            VariantLaunchSettings variantSettings,
                                                            unsigned int totalRedNumber,
                                                            PREC relTol,
                                                            PREC absTol);
#undef PREC

